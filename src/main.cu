#include "struct.cuh"
#include "astroio.h"
#include <cstdlib>
#include <vector>
#include <string>
#include <filesystem>
#include "optionsparser.h"
extern std::string directorypath;
extern std::string outputfilename;
void listfiles(std::string directorypath,std::vector<std::string> &list_of_files){
    for (const auto& entry : std::filesystem::directory_iterator(directorypath)) {
        std::filesystem::path outfilename = entry.path();
        std::string outfilename_str = outfilename.string();
        //std::cout<<outfilename_str<<std::endl;
        list_of_files.push_back(outfilename_str);
    }
}

int main(int argc, char ** argv){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    hipMemPool_t memPool;
    size_t free, total;
    hipMemGetInfo(&free,&total);
    std::cout<<"free="<<free<<",total="<<total<<std::endl;
    std::vector<std::string> files;
    optionparser(argc, argv);
    if (directorypath.empty()){
        std::cout<<"Input directory has not been mentioned"<<std::endl;
        exit(1);
    }
    listfiles(directorypath,files);
    std::stable_sort(files.begin(), files.end());
    astrojpg_rgb_<Npp8u> image1(files[0]);
    std::cout<<image1.nppinputimage.width()<<","<<image1.nppinputimage.height();
    
    
    std::cout<<files[1]<<std::endl;
    //for (auto filename: files) std::cout<<filename<<std::endl;
    image1.getgreyimage(); //get the grey image 
    Npp8u* maxbuffer;
    Npp8u* sumbuffer;
    size_t  maxbufferhostsize;
    
    NppiSize osizeROI={(int)image1.nppgreyimage.width(),(int)image1.nppgreyimage.height()};
    
    nppiMaxIndxGetBufferHostSize_8u_C1R(osizeROI, &maxbufferhostsize);

    hipMalloc((void**)&maxbuffer,maxbufferhostsize);

    image1.getmaxpixel(image1.nppgreyimage,image1.maxpixelposition,maxbuffer);
    std::cout<<"Maximum pixel: x="<<image1.maxpixelposition.x<<",y="<<image1.maxpixelposition.y<<std::endl;

    const Npp8u threshold=40;
    image1.getsignalimage(image1.nppgreyimage,threshold);

    saveastro<Npp8u,1>(image1.nppgreyimage,"greyimage.jpg");
    saveastro<Npp8u,1>(image1.signalimage,"signalimage.jpg");
    /* We then need to create the data that will be use for the convolution*/
    int squaresize=111; //square size 

    size_t  sumbufferhostsize;
    NppiSize omaskROI={squaresize,squaresize};
    nppiSumGetBufferHostSize_32f_C1R(omaskROI, &sumbufferhostsize);
    hipMalloc((void**)&sumbuffer,sumbufferhostsize);
    std::cout<<"Am I here"<<std::endl;
    hipDeviceSynchronize();
    image1.createROIdata(squaresize);
    saveastro<Npp32s,1>(image1.maskimage,"mask.jpg");
    
    std::cout<<"Am I here 4"<<std::endl;
    std::cout<<"FINAL COUNTDOWN"<<std::endl;
    
    /*Create a new function for mosaicing the stuff*/
    int differencex,differencey;
    
    astrojpg_rgb_<Npp32f> imagetotal(files[1]);
    imagetotal.getgreyimage();
    imagetotal.getsignalimage(imagetotal.nppgreyimage,threshold);
    imagetotal.Correlationimage(image1.maskimage,sumbuffer);
    imagetotal.getmaxpixel(imagetotal.correlationimage,imagetotal.maxcorrposition,maxbuffer);
    hipDeviceSynchronize();
    std::cout<<"am i here now"<<std::endl;
    hipMemGetInfo(&free,&total);
    std::cout<<"free="<<free<<",total="<<total<<std::endl;
    int i=0;
    for (std::string file : files ){
        std::cout<<"filename="<<file<<std::endl;
        astrojpg_rgb_<Npp8u> iterimage(file);
        
        iterimage.getgreyimage();
        
        iterimage.getsignalimage(iterimage.nppgreyimage,threshold);
        //std::string itersignal="itersignal"+std::to_string(i)+".jpg";
        //saveastro<Npp8u,1>(iterimage.signalimage,itersignal);
        iterimage.Correlationimage(image1.maskimage,sumbuffer);
        //std::string itercorr="itercorr"+std::to_string(i)+".jpg";
        //saveastro<Npp8u,1>(iterimage.correlationimage,itercorr);
        iterimage.getmaxpixel(iterimage.correlationimage,iterimage.maxcorrposition,maxbuffer);
        
        differencex=iterimage.maxcorrposition.x-imagetotal.maxcorrposition.x;
        differencey=iterimage.maxcorrposition.y-imagetotal.maxcorrposition.y;
        cv::Point_<int> offsetposition={differencex,differencey};
        std::cout<<"totalwidth="<<imagetotal.nppinputimage.width()<<","<<imagetotal.nppinputimage.height()<<std::endl;
    
        std::cout<<"diff="<<differencex<<","<<differencey<<std::endl;
        std::cout<<"Am I now"<<std::endl;
        imagetotal.stackimage(iterimage);
        std::cout<<imagetotal.nppinputimage.width()<<","<<imagetotal.nppinputimage.height()<<std::endl;
        hipDeviceSynchronize();
        //std::string iterstack="finalresult_"+std::to_string(i)+".jpg";
        //saveastro<Npp32f,3>(imagetotal.nppinputimage,iterstack);
        //std::string iterexp="finalexp_"+std::to_string(i)+".jpg";
        //saveastro<Npp32f,1>(imagetotal.exposuremap,iterexp);
        i++;
    }
    
    //std::cout<<"diff="<<imagetotal.maxcorrposition.x<<","<<imagetotal.maxcorrposition.y<<std::endl;
    //cv::Point_<int> offsetposition={differencex,differencey};
    //imagetotal.stackimage(image3,offsetposition);
    saveastro<Npp32f,3>(imagetotal.nppinputimage,outputfilename);
    saveastro<Npp32f,1>(imagetotal.exposuremap,"finalresultexp.jpg");
    return 0;
}