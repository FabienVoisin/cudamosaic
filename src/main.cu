#include "struct.cuh"
#include "astroio.h"
#include <cstdlib>
#include <vector>
#include <string>
#include <filesystem>
#include "optionsparser.h"
/*Extern variables */
extern std::string directorypath;
extern std::string outputfilename;

/*global variable : NPP buffers */
Npp8u *maxbuffer;
Npp8u *sumbuffer;
int squaresize;
const Npp8u threshold=40;

void setupmaxbuffer(astrojpg_rgb_<Npp8u> &image1){
    size_t  maxbufferhostsize;
    NppiSize osizeROI={(int)image1.nppgreyimage.width(),(int)image1.nppgreyimage.height()};
    nppiMaxIndxGetBufferHostSize_8u_C1R(osizeROI, &maxbufferhostsize);
    hipMalloc((void**)&maxbuffer,maxbufferhostsize);
}

void setupsumbuffer(int squaresize){
    size_t  sumbufferhostsize;
    NppiSize osumROI={squaresize,squaresize};
    nppiSumGetBufferHostSize_32f_C1R(osumROI, &sumbufferhostsize);
    hipMalloc((void**)&sumbuffer,sumbufferhostsize);
}

void initfirstimage(astrojpg_rgb_<Npp8u> &image1){
    setupmaxbuffer(image1);
    setupsumbuffer(squaresize);
    image1.getgreyimage(); 
    image1.getsignalimage(image1.nppgreyimage,threshold);
    image1.getmaxpixel(image1.nppgreyimage,image1.maxpixelposition,maxbuffer);
    hipDeviceSynchronize();
    image1.createROIdata(squaresize);

}

void mosaicimages(std::vector<std::string> &files, astrojpg_rgb_<Npp32f> &imagetotal, astrojpg_rgb_<Npp8u> &image1){
    
    int differencex,differencey;
    imagetotal.getgreyimage();
    imagetotal.getsignalimage(imagetotal.nppgreyimage,threshold);
    imagetotal.Correlationimage(image1.maskimage,sumbuffer);
    imagetotal.getmaxpixel(imagetotal.correlationimage,imagetotal.maxcorrposition,maxbuffer);

    for (std::string file : files ){
        std::cout<<"filename="<<file<<std::endl;
        astrojpg_rgb_<Npp8u> iterimage(file);
        
        iterimage.getgreyimage();
        
        iterimage.getsignalimage(iterimage.nppgreyimage,threshold);
        iterimage.Correlationimage(image1.maskimage,sumbuffer);
        iterimage.getmaxpixel(iterimage.correlationimage,iterimage.maxcorrposition,maxbuffer);
        
        differencex=iterimage.maxcorrposition.x-imagetotal.maxcorrposition.x;
        differencey=iterimage.maxcorrposition.y-imagetotal.maxcorrposition.y;
        cv::Point_<int> offsetposition={differencex,differencey};
        imagetotal.stackimage(iterimage);
        std::cout<<imagetotal.nppinputimage.width()<<","<<imagetotal.nppinputimage.height()<<std::endl;
        hipDeviceSynchronize();
        std::cout<<"filename="<<file<<std::endl;
    }
}

void normaliseimage(astrojpg_rgb_<Npp32f> &imagetotal){
    imagetotal.normaliseimage<Npp32f,3>(imagetotal.nppinputimage);
    imagetotal.normaliseimage<Npp32f,1>(imagetotal.nppgreyimage);
}

void outputmosaic(astrojpg_rgb_<Npp32f> &imagetotal){
    saveastro<Npp32f,3>(imagetotal.nppinputimage,outputfilename);    
    saveastro<Npp32f,3>(imagetotal.exposuremap,"finalresultexp.jpg");
}

int main(int argc, char **argv){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    hipMemPool_t memPool;
    size_t free, total;
    hipMemGetInfo(&free,&total);
    std::cout<<"free="<<free<<",total="<<total<<std::endl;
    std::vector<std::string> files;
    
    optionparser(argc, argv);
    if (directorypath.empty()){
        std::cout<<"Input directory has not been mentioned"<<std::endl;
        exit(1);
    }
    listfiles(directorypath,files);
    std::stable_sort(files.begin(), files.end());
    squaresize=111;
    astrojpg_rgb_<Npp8u> image1(files[0]);
    initfirstimage(image1);

    astrojpg_rgb_<Npp32f> imagetotal(files[1]);
    mosaicimages(files,imagetotal,image1);
    normaliseimage(imagetotal);
    outputmosaic(imagetotal);

    
    return 0;
}