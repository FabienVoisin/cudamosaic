#include "struct.cuh"
#include "astroio.h"

int main(){
    astrojpg_8u_rgb image1("Orion/orion_79.jpg");
    image1.getgreyimage(); //get the grey image 
    Npp8u* maxbuffer;
    Npp8u* sumbuffer;
    size_t  maxbufferhostsize;
    
    NppiSize osizeROI={(int)image1.nppgreyimage.width(),(int)image1.nppgreyimage.height()};
    
    nppiMaxIndxGetBufferHostSize_8u_C1R(osizeROI, &maxbufferhostsize);

    hipMalloc((void**)&maxbuffer,maxbufferhostsize);

    image1.getmaxpixel(image1.nppgreyimage,image1.maxpixelposition,maxbuffer);
    std::cout<<"Maximum pixel: x="<<image1.maxpixelposition.x<<",y="<<image1.maxpixelposition.y<<std::endl;

    const Npp8u threshold=40;
    image1.getsignalimage(image1.nppgreyimage,threshold);

    saveastro<Npp8u,1>(image1.nppgreyimage,"greyimage.jpg");
    saveastro<Npp8u,1>(image1.signalimage,"signalimage.jpg");
    /* We then need to create the data that will be use for the convolution*/
    int squaresize=111; //square size 

    size_t  sumbufferhostsize;
    NppiSize omaskROI={squaresize,squaresize};
    nppiSumGetBufferHostSize_32f_C1R(omaskROI, &sumbufferhostsize);
    hipMalloc((void**)&sumbuffer,sumbufferhostsize);
    std::cout<<"Am I here"<<std::endl;
    hipDeviceSynchronize();
    image1.createROIdata(squaresize);
    saveastro<Npp32s,1>(image1.maskimage,"mask.jpg");
    //image1.Correlationimage(image1.maskimage,sumbuffer);
    //saveastro<Npp8u,1>(image1.correlationimage,"autocorrelation1.jpg");
    //image1.getmaxpixel(image1.correlationimage,image1.maxcorrposition,maxbuffer);
    //std::cout<<"Maximum corr pixel: x="<<image1.maxcorrposition.x<<",y="<<image1.maxcorrposition.y<<std::endl;

    /*We need to load a new  image */
    std::cout<<"Am I here 2"<<std::endl;
    astrojpg_8u_rgb image2("Orion/orion_1.jpg");
    image2.getgreyimage();
    image2.getsignalimage(image2.nppgreyimage,threshold);
    /*Create three new images, the correlation image, the exposure map and the new combinbed image*/
    //npp::ImageNPP_8u_C1 correlation(image2.nppgreyimage.size());
    image2.Correlationimage(image1.maskimage,sumbuffer);
    std::cout<<"FINAL COUNTDOWN"<<std::endl;
    saveastro<Npp8u,1>(image2.correlationimage,"correlationexample.jpg");
    image2.getmaxpixel(image2.correlationimage,image2.maxcorrposition,maxbuffer);
    std::cout<<"Maximum corr pixel: x="<<image2.maxcorrposition.x<<",y="<<image2.maxcorrposition.y<<std::endl;
    astrojpg_8u_rgb image3("Orion/orion_82.jpg");
    image3.getgreyimage();
    image3.getsignalimage(image3.nppgreyimage,threshold);
    image3.Correlationimage(image1.maskimage,sumbuffer);
    saveastro<Npp8u,1>(image3.correlationimage,"correlationexample2.jpg");
    image3.getmaxpixel(image3.correlationimage,image3.maxcorrposition,maxbuffer);
    std::cout<<"Maximum corr pixel: x="<<image3.maxcorrposition.x<<",y="<<image3.maxcorrposition.y<<std::endl;

    /*Create a new function for mosaicing the stuff*/
    unsigned int differencex,differencey;
    differencex=image3.maxcorrposition.x-image2.maxcorrposition.x;
    differencey=image3.maxcorrposition.y-image2.maxcorrposition.y;
    astrojpg_8u_rgb imagetotal(image1.signalimage.width()+differencex,image1.signalimage.height()+differencey);
    npp::ImageNPP_8u_C1 imageexposure(image1.signalimage.width()+differencex,image1.signalimage.height()+differencey);
    
    

    return 0;
}