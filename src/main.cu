#include "struct.cuh"
#include "gpufuncs.cuh"
#include "astroio.h"

int main(){
    astrojpg_8u_rgb image1("Orion/orion_1.jpg");
    image1.getgreyimage(); //get the grey image 
    Npp8u* maxbuffer;
    Npp8u* sumbuffer;
    size_t  maxbufferhostsize;
    
    NppiSize osizeROI={(int)image1.nppgreyimage.width(),(int)image1.nppgreyimage.height()};
    
    nppiMaxIndxGetBufferHostSize_8u_C1R(osizeROI, &maxbufferhostsize);

    hipMalloc((void**)&maxbuffer,maxbufferhostsize);

    image1.getmaxpixel(image1.nppgreyimage,maxbuffer);
    std::cout<<"Maximum pixel: x="<<image1.maxpixelposition.x<<",y="<<image1.maxpixelposition.y<<std::endl;

    const Npp8u threshold=10;
    image1.getsignalimage(threshold);

    saveastro<Npp8u,1>(image1.nppgreyimage,"greyimage.jpg");
    saveastro<Npp8u,1>(image1.signalimage,"signalimage.jpg");
    /* We then need to create the data that will be use for the convolution*/
    int squaresize=111; //square size 
    npp::ImageNPP_32s_C1 maskimage(squaresize,squaresize);

    size_t  sumbufferhostsize;
    NppiSize omaskROI={squaresize,squaresize};
    nppiSumGetBufferHostSize_32f_C1R(omaskROI, &sumbufferhostsize);
    hipMalloc((void**)&sumbuffer,sumbufferhostsize);
    std::cout<<"Am I here"<<std::endl;
    hipDeviceSynchronize();
    CreateROIdata(image1.signalimage,image1.maxpixelposition,squaresize,maskimage);
    saveastro<Npp32s,1>(maskimage,"mask.jpg");
    /*We need to load a new  image */
    std::cout<<"Am I here 2"<<std::endl;
    astrojpg_8u_rgb image2("Orion/orion_48.jpg");
    image2.getgreyimage();
    image2.getsignalimage(threshold);
    /*Create three new images, the correlation image, the exposure map and the new combinbed image*/
    npp::ImageNPP_8u_C1 correlation(image2.nppgreyimage.size());
    Correlationimage(image2.signalimage,maskimage,sumbuffer,correlation);
    std::cout<<"FINAL COUNTDOWN"<<std::endl;
    saveastro<Npp8u,1>(correlation,"correlationexample.jpg");

    
    return 0;
}