#include "struct.cuh"
#include "astroio.h"

int main(){
    astrojpg_8u_rgb image1("Orion/orion_1.jpg");
    image1.getgreyimage(); //get the grey image 
    Npp8u* maxbuffer;
    Npp8u* sumbuffer;
    size_t  maxbufferhostsize;
    
    NppiSize osizeROI={(int)image1.nppgreyimage.width(),(int)image1.nppgreyimage.height()};
    
    nppiMaxIndxGetBufferHostSize_8u_C1R(osizeROI, &maxbufferhostsize);

    hipMalloc((void**)&maxbuffer,maxbufferhostsize);

    image1.getmaxpixel(image1.nppgreyimage,maxbuffer);
    std::cout<<"Maximum pixel: x="<<image1.maxpixelposition.x<<",y="<<image1.maxpixelposition.y<<std::endl;

    const Npp8u threshold=40;
    image1.getsignalimage(threshold);

    saveastro<Npp8u,1>(image1.nppgreyimage,"greyimage.jpg");
    saveastro<Npp8u,1>(image1.signalimage,"signalimage.jpg");
    /* We then need to create the data that will be use for the convolution*/
    int squaresize=111; //square size 

    size_t  sumbufferhostsize;
    NppiSize omaskROI={squaresize,squaresize};
    nppiSumGetBufferHostSize_32f_C1R(omaskROI, &sumbufferhostsize);
    hipMalloc((void**)&sumbuffer,sumbufferhostsize);
    std::cout<<"Am I here"<<std::endl;
    hipDeviceSynchronize();
    image1.createROIdata(squaresize);
    saveastro<Npp32s,1>(image1.maskimage,"mask.jpg");
    /*We need to load a new  image */
    std::cout<<"Am I here 2"<<std::endl;
    astrojpg_8u_rgb image2("Orion/orion_48.jpg");
    image2.getgreyimage();
    image2.getsignalimage(threshold);
    /*Create three new images, the correlation image, the exposure map and the new combinbed image*/
    npp::ImageNPP_8u_C1 correlation(image2.nppgreyimage.size());
    image2.Correlationimage(image1.maskimage,sumbuffer);
    std::cout<<"FINAL COUNTDOWN"<<std::endl;
    saveastro<Npp8u,1>(image2.correlationimage,"correlationexample.jpg");

    
    return 0;
}