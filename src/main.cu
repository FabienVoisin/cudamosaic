#include "struct.cuh"

int main(){
    astrojpg_8u_rgb image1("Orion/orion_1.jpg");
    image1.getgreyimage(); //get the grey image 
    Npp8u* maxbuffer;
    size_t  maxbufferhostsize;
    
    NppiSize osizeROI={(int)image1.nppgreyimage.width(),(int)image1.nppgreyimage.height()};
    
    nppiMaxIndxGetBufferHostSize_8u_C1R(osizeROI, &maxbufferhostsize);

    hipMalloc((void**)&maxbuffer,maxbufferhostsize);

    image1.getmaxpixel(image1.nppgreyimage,maxbuffer);
    std::cout<<"Maximum pixel: x="<<image1.maxpixelposition.x<<",y="<<image1.maxpixelposition.y;

    const Npp8u threshold=10;
    image1.getsignalimage(threshold);
    saveastro<Npp8u,1>(image1.signalimage,"signalimage.jpg");

    
    return 0;
}