/*Author : Fabien Voisin*/
#include <iostream>
#include <Common/UtilNPP/ImagesCPU.h>
#include <Common/UtilNPP/ImagesNPP.h>
#include <nppdefs.h>
//#include "/mnt/sdd/cuda-samples/Common/UtilNPP/ImageIO.h"
//#include <opencv2\opencv.hpp> 
#include <opencv2/imgcodecs.hpp>
#include <opencv2/core.hpp>
#include <vector>

int main(int argc, char **argv){
cv::Mat img = imread("data/20241209_230358.jpg", cv::IMREAD_COLOR);

int imagewidth=img.size().width;
int imageheight=img.size().height;
//int bytes=img.imageSize;
//ROWS and COLUMNS ARE NOT IN BYTES 
//RGB is 8u thus 1 byte per CHANNELS

std::cout<<img.total()<<std::endl;
std::cout<<img.step1()<<":"<<img.size().width*img.channels()<<","<<img.step[1]<<","<<img.step[2]<<std::endl;
std::cout<<img.rows<<","<<img.cols<<std::endl;
//unsigned char *dSrc, *dDst;
//dSrc=img.data;
unsigned char *input = (unsigned char*)(img.data);
int i,j,r,g,b;
    /*for(int i = 0;i < img.rows  ;i++){
			for(int j = 0;j < img.cols*3 ;j+=3){
                b = img.data[img.step[0] * i + j*img.step[1] ] ;
                g = img.data[img.step[0] * i + j*img.step[1] + 1];
                r = img.data[img.step[0] * i + j*img.step[1] + 2];
                //img.at<cv::Vec3b>(i,j)[0];
                if(i>1600 && i< 1700 && b> 100) std::cout<<"("<<i<<","<<j/3<<"):"<<"r="<<r<<",g="<<g<<",b="<<b<<std::endl;
                //if(i>1600 && i< 1700) std::cout<<"("<<i<<","<<j/3<<"):"<<"b="<<img.at<cv::Vec3b>(i,j)[0]<<",g="<<img.at<cv::Vec3b>(i,j)[1]<<",b="<<img.at<cv::Vec3b>(i,j)[2]<<std::endl;
            }
        }*/

//memcpy( &dst[dstIdx], &src[srcIdx], numElementsToCopy * sizeof( Element ) );
std::cout<<"end"<<std::endl;
//The next step is to understand the data structure of the image on a per channel basis.
//std::cout<<dSrc[8]<<std::endl;

//hipMalloc<unsigned char>(&dSrc,bytes);
npp::ImageCPU_8u_C3 inputfile((unsigned int)img.size().width,(unsigned int)img.size().height);
const Npp8u *pDstLine=inputfile.data();
unsigned int nDstPitch = inputfile.pitch();
for (size_t i=0;i<img.rows;i++){
    //memcpy(inputfile.data(0,i),&input[img.rows*i],img.step1()*sizeof(Npp8u));
    memcpy(inputfile.data(0,i),&input[img.step[0]*i],img.step1());
    pDstLine+=nDstPitch;
}

for(int i=0;i<inputfile.height();i++){
    for(int j=0;j<inputfile.width();j++){
        inputfile.pixels(j,i)[0].x+=10;
        inputfile.pixels(j,i)[1].y+=0;
        inputfile.pixels(j,i)[2].z+=50;
        
    }

}

// This will create a specific pitch 
//std::cout<<"value15="<<(unsigned char)inputfile.pixels(1,5)[0].x<<std::endl;
npp::ImageCPU_8u_C3 osrchost; // first input data;

int linestep;
std::cout<<"test"<<std::endl;



int devicewidth=linestep;
int deviceheight=img.size().height;
NppiSize osizeROI={devicewidth,deviceheight};


const Npp8u values[3]={1,2,3};


cv::Mat outputimg((int)inputfile.height(),(int)inputfile.width(),CV_8UC3,(void *)inputfile.data(),(size_t)inputfile.pitch());
std::cout<<outputimg.step1()<<":"<<outputimg.size().width*outputimg.channels()<<","<<outputimg.step[1]<<","<<outputimg.step[2]<<std::endl;

;
unsigned char *output = (unsigned char*)(outputimg.data);
/*for (size_t i=0;i<outputimg.rows;i++){
    memcpy(&output[outputimg.cols*i],inputfile.data(0,i),inputfile.pitch());
}*/

bool check=imwrite("newimg2.jpg",outputimg);
std::cout<<outputimg.total()<<std::endl;

for(int i = 0;i < outputimg.rows  ;i++){
    for(int j = 0;j < outputimg.cols ;j+=3){
        b = output[outputimg.cols * i + j ] ;
        g = output[outputimg.cols * i + j + 1];
        r = output[outputimg.cols * i + j + 2];
        //if(b>16) std::cout<<"r="<<r<<",g="<<g<<",b="<<b<<std::endl;
    }
   // std::cout<<i<<","<<j<<std::endl;
}
//nppiAddC_8u_C3IRSfs(values,inputfile,linestep,osizeROI, 0);
/*Set outcome to host image*/
//npp::ImageCPU_8u_C3 ohostdest(&inputfile.size());

/*odevgrey.copyTo(ohostdest.data(),oHosdest.pitch());

SaveImage("testgrey.jpg",ohostdest);*/





return 0;

}