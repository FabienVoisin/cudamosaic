#include "hip/hip_runtime.h"
#include "gpufuncs.cuh"
/*CreateROIdata  will create an array in the region of interest surrounding the max values of the greyimage
 which will be needed for the autoconvolution*/

npp::ImageCPU_32s_C1 &CreateROIdata(npp::ImageNPP_8u_C1 &greyimage,cv::Point_<int> &maxpixelposition, int squaresize){
NppiSize osizeROI={squaresize,squaresize}; //Setup the osizeROI
/*The two values will help find the appropriate address of the first value*/
int initpositionx=maxpixelposition.x-squaresize/2+1; 
int initpositiony=maxpixelposition.y-squaresize/2+1;
npp::ImageCPU_8u_C1 outputROIimage(squaresize,squaresize);
npp::ImageCPU_8u_C1 outputmirrorimage(squaresize,squaresize);
npp::ImageCPU_32s_C1 outputfinalimage(squaresize,squaresize);
hipError_t eResult;
eResult=hipMemcpy2D(outputROIimage.data(),outputROIimage.pitch(),greyimage.data(initpositionx,initpositiony),greyimage.pitch(),outputROIimage.width()*sizeof(Npp8u),outputROIimage.height(),hipMemcpyDeviceToHost);
NPP_ASSERT(hipSuccess == eResult);
/*nppiMirror will flip the image so that the last values become the first, necessary for the convolution*/
nppiMirror_8u_C1R(outputROIimage.data(),(int)outputROIimage.pitch(), outputmirrorimage.data(),(int)outputmirrorimage.pitch(), osizeROI, NPP_BOTH_AXIS);

/*Finally we need to convert to 32s*/
nppiConvert_8u32s_C1R(outputmirrorimage.data(),outputmirrorimage.pitch(),outputfinalimage.data(),outputfinalimage.pitch(),osizeROI);


return outputfinalimage;

}


npp::ImageNPP_8u_C1 &Correlationimage(npp::ImageNPP_8u_C1 &referenceimage,npp::ImageCPU_32s_C1 &maskimage){
    
    npp::ImageNPP_8u_C1 outputimage(referenceimage.width(),referenceimage.height());
    NppiSize osizeROI={(int)referenceimage.width(),(int)referenceimage.height()};
    NppiSize okernelSize={(int)maskimage.width(),(int)maskimage.height()};
    NppiPoint oAnchor={(int)(maskimage.width()/2),(int)(maskimage.height()/2)};
    Npp32s ndivisor=maskimage.width()*maskimage.height();
    NppStatus status=nppiFilter_8u_C1R(referenceimage.data(), referenceimage.pitch(), outputimage.data(), outputimage.pitch(),osizeROI, maskimage.data(), okernelSize, oAnchor, ndivisor);

    return outputimage;
}