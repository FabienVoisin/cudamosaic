/*Author : Fabien Voisin*/
#include <iostream>
#include <Common/UtilNPP/ImagesCPU.h>
#include <Common/UtilNPP/ImagesNPP.h>
#include <nppdefs.h>
//#include "/mnt/sdd/cuda-samples/Common/UtilNPP/ImageIO.h"
//#include <opencv2\opencv.hpp> 
#include <opencv2/imgcodecs.hpp>
#include <opencv2/core.hpp>
#include <vector>

int main(int argc, char **argv){
cv::Mat img = imread("data/20241209_230358.jpg", cv::IMREAD_COLOR);

int imagewidth=img.size().width;
int imageheight=img.size().height;
//int bytes=img.imageSize;
//ROWS and COLUMNS ARE NOT IN BYTES 
//RGB is 8u thus 1 byte per CHANNELS

std::cout<<img.total()<<std::endl;
std::cout<<img.step1()<<":"<<img.size().width*img.channels()<<","<<img.step[1]<<","<<img.step[2]<<std::endl;
std::cout<<img.rows<<","<<img.cols<<std::endl;
//unsigned char *dSrc, *dDst;
//dSrc=img.data;
unsigned char *input = (unsigned char*)(img.data);
int i,j,r,g,b;
    /*for(int i = 0;i < img.rows  ;i++){
			for(int j = 0;j < img.cols*3 ;j+=3){
                b = img.data[img.step[0] * i + j*img.step[1] ] ;
                g = img.data[img.step[0] * i + j*img.step[1] + 1];
                r = img.data[img.step[0] * i + j*img.step[1] + 2];
                //img.at<cv::Vec3b>(i,j)[0];
                if(i>1600 && i< 1700 && b> 100) std::cout<<"("<<i<<","<<j/3<<"):"<<"r="<<r<<",g="<<g<<",b="<<b<<std::endl;
                //if(i>1600 && i< 1700) std::cout<<"("<<i<<","<<j/3<<"):"<<"b="<<img.at<cv::Vec3b>(i,j)[0]<<",g="<<img.at<cv::Vec3b>(i,j)[1]<<",b="<<img.at<cv::Vec3b>(i,j)[2]<<std::endl;
            }
        }*/

//memcpy( &dst[dstIdx], &src[srcIdx], numElementsToCopy * sizeof( Element ) );
std::cout<<"end"<<std::endl;
//The next step is to understand the data structure of the image on a per channel basis.
//std::cout<<dSrc[8]<<std::endl;

//hipMalloc<unsigned char>(&dSrc,bytes);
npp::ImageCPU_8u_C3 inputfile((unsigned int)img.size().width,(unsigned int)img.size().height);
for (size_t i=0;i<img.rows;i++){
    //memcpy(inputfile.data(0,i),&input[img.rows*i],img.step1()*sizeof(Npp8u));
    memcpy(inputfile.data(0,i),&input[img.step[0]*i],img.step1());
}


/*for(int i=0;i<inputfile.height();i++){
    for(int j=0;j<inputfile.width();j++){
        inputfile.pixels(j,i)[0].x+=0;
        inputfile.pixels(j,i)[1].y+=0;
        inputfile.pixels(j,i)[2].z+=0;
        
    }

}*/


int linestep;
std::cout<<"test"<<std::endl;

/*Now create a NPP image output*/
npp::ImageNPP_8u_C3 nppinputfile(inputfile);
/*Create a grey image*/

npp::ImageNPP_8u_C1 nppgreyfile(nppinputfile.width(),nppinputfile.height());



std::cout<<"newtest"<<std::endl;


int devicewidth=linestep;
int deviceheight=img.size().height;

NppiSize osizeROI;
osizeROI.width=(int)nppinputfile.width();
osizeROI.height=(int)nppinputfile.height();

std::cout<<"nppinputfile:width="<<(int)nppinputfile.width()<<",pitch="<<(int)nppinputfile.pitch()<<std::endl;
nppiRGBToGray_8u_C3C1R(nppinputfile.data(), nppinputfile.pitch(), nppgreyfile.data(), nppgreyfile.pitch(),osizeROI);

Npp8u* maxbuffer;
size_t  maxbufferhostsize;
int *positionx;
int *positiony;
cv::Point_<int> hostmaxpixelposition;
Npp8u *nppmaxvalues;

nppiMaxIndxGetBufferHostSize_8u_C1R(osizeROI, &maxbufferhostsize);
hipMalloc((void**)&maxbuffer,maxbufferhostsize);
hipMalloc((void**)&nppmaxvalues, sizeof(Npp8u) * 1);
hipMalloc((void**)&positionx,sizeof(int));
hipMalloc((void**)&positiony,sizeof(int));

nppiMaxIndx_8u_C1R(nppgreyfile.data(), nppgreyfile.pitch(),osizeROI,maxbuffer, nppmaxvalues, positionx, positiony);
Npp8u hostmaxvalues;
hipMemcpy(&hostmaxvalues,nppmaxvalues,sizeof(Npp8u) * 1,hipMemcpyDeviceToHost);
hipMemcpy(&(hostmaxpixelposition.x),positionx,sizeof(int),hipMemcpyDeviceToHost);
hipMemcpy(&(hostmaxpixelposition.y),positiony,sizeof(int),hipMemcpyDeviceToHost);
std::cout<<"maxpixel(x="<<hostmaxpixelposition.x<<",y="<<hostmaxpixelposition.y<<")="<<(int)hostmaxvalues<<std::endl;

npp::ImageNPP_8u_C1 nppdestfile(nppinputfile.width(),nppinputfile.height());
const Npp8u threshold=10;
std::cout<<"newtest2"<<std::endl;
//nppiThreshold_8u_C3R(const Npp8u *pSrc, int nSrcStep, Npp8u *pDst, int nDstStep, NppiSize oSizeROI, const Npp8u rThresholds[3], NppCmpOp eComparisonOperation);
//NppStatus status=nppiThreshold_8u_C3IR(nppthreshold.data(),nppthreshold.pitch(), osizeROI, threshold, NPP_CMP_GREATER_EQ);
nppiCompareC_8u_C1R(nppgreyfile.data(),(int)nppgreyfile.pitch(), threshold, nppdestfile.data(),(int)nppdestfile.pitch(),osizeROI,NPP_CMP_GREATER_EQ);
//std::cout<<"status="<<status<<std::endl;

npp::ImageCPU_8u_C1 oHostdest(nppdestfile.size());
/*Npw we need to get the value back to the inputfile */
nppdestfile.copyTo(oHostdest.data(),oHostdest.pitch());

//nppinputfile.copyTo(oHostdest.data(),oHostdest.pitch());


std::cout<<"newtest4"<<std::endl;

cv::Mat outputimg((int)oHostdest.height(),(int)oHostdest.width(),CV_8UC1,(void *)oHostdest.data(),(size_t)oHostdest.pitch());
//cv::Mat outputimg((int)oHostdest.height(),(int)oHostdest.width(),CV_8UC1);
std::cout<<outputimg.step1()<<":"<<oHostdest.pitch()<<","<<outputimg.step[0]<<","<<std::endl;

;
unsigned char *output = (unsigned char*)(outputimg.data);
/*for (size_t i=0;i<outputimg.rows;i++){
    //std::cout<<"iteration="<<i<<std::endl;
    memcpy(&output[img.step[0]*i],oHostdest.data(0,i),oHostdest.pitch());
    //memcpy(&output[img.step[0]*i],oHostdest.data(0,i),outputimg.step[0]);
}*/
std::cout<<"final"<<std::endl;
bool check=imwrite("newimg4.jpg",outputimg);
std::cout<<outputimg.total()<<std::endl;

/*for(int i = 0;i < outputimg.rows  ;i++){
    for(int j = 0;j < outputimg.cols*3 ;j+=3){
        b = output[img.step[0] * i + j*img.step[1] ] ;
        g = output[img.step[0] * i + j*img.step[1] + 1];
        r = output[img.step[0] * i + j*img.step[1] + 2];
        std::cout<<"r="<<r<<",g="<<g<<",b="<<b<<std::endl;
    }
   // std::cout<<i<<","<<j<<std::endl;
}*/
//nppiAddC_8u_C3IRSfs(values,inputfile,linestep,osizeROI, 0);
/*Set outcome to host image*/
//npp::ImageCPU_8u_C3 ohostdest(&inputfile.size());

/*odevgrey.copyTo(ohostdest.data(),oHosdest.pitch());

SaveImage("testgrey.jpg",ohostdest);*/





return 0;

}